#include "hip/hip_runtime.h"
#include <iostream>
#include <limits.h>
#include <stdlib.h>
#include <ctime>
#include <sstream>
#include <string>

#include "gpu_hashtable.hpp"

#define BLOCKSIZE 1024

typedef struct {
	int key;
	int value;
} Pair;

typedef struct {
	int *size;
	int *numElem;
	Pair *pairs;
} HashTable;

HashTable hashTable;

/* INIT HASH
 */
GpuHashTable::GpuHashTable(int size) {
	hipMalloc(&hashTable.size, sizeof(int));
	hipMalloc(&hashTable.pairs, size * sizeof(Pair));
	hipMalloc(&hashTable.numElem, sizeof(int));

	hipMemset(hashTable.numElem, 0, sizeof(int));
	hipMemset(hashTable.pairs, 0, size * sizeof(Pair));
	hipMemcpy(hashTable.size, &size, sizeof(int), hipMemcpyHostToDevice);	
}

/* DESTROY HASH
 */
GpuHashTable::~GpuHashTable() {
	hipFree(hashTable.pairs);
	hipFree(hashTable.size);
	hipFree(hashTable.numElem);
}

__global__ void resize(Pair *newPairs, Pair *oldPairs, int *size, int *numElem, int *oldSize) {
	int keyToInsert = blockIdx.x * blockDim.x + threadIdx.x;
	if (keyToInsert >= *oldSize)
		return;

	int key = oldPairs[keyToInsert].key;
	if (key == 0)
		return;

	int position = hash1(key, *size);
	int index = position;
	int free = 0;

	/* Search for an empty position. */
	while (atomicCAS(&(newPairs[index].key), free, key) != 0) {
		index++;
		if (index == (*size))
			index = 0;
	}

	atomicAdd(numElem, 1);
	newPairs[index].value = oldPairs[keyToInsert].value;
}

/* RESHAPE HASH
 */
void GpuHashTable::reshape(int numBucketsReshape) {
	Pair *devicePairs;
	int hostOldSize;
	int *deviceOldSize;

	hipMalloc(&deviceOldSize, sizeof(int));
	hipMalloc(&devicePairs, numBucketsReshape * sizeof(Pair));

	hipMemcpy(deviceOldSize, hashTable.size, sizeof(int), hipMemcpyDeviceToDevice);
	hipMemcpy(&hostOldSize, hashTable.size, sizeof(int), hipMemcpyDeviceToHost);

	hipMemset(hashTable.numElem, 0, sizeof(int));
	hipMemcpy(hashTable.size, &numBucketsReshape, sizeof(int), hipMemcpyHostToDevice);	

	/* Compute number of blocks */
	int blockNum = hostOldSize / BLOCKSIZE;
	if (blockNum * BLOCKSIZE < hostOldSize)
		blockNum++;

	resize<<<blockNum, BLOCKSIZE>>>(devicePairs, hashTable.pairs,
			hashTable.size, hashTable.numElem, deviceOldSize);

	hipDeviceSynchronize();

	hipFree(hashTable.pairs);
	hashTable.pairs = devicePairs;
}

__global__ void insert(int *keys, int *values, Pair *pairs, int *size,
		int *numElem, int *result, int numKeys) {
	/* Get position in HashTable for inertion */
	int keyToInsert = blockIdx.x * blockDim.x + threadIdx.x;
	if (keyToInsert >= numKeys)
		return;

	if (keys[keyToInsert] <= 0 || values[keyToInsert] <= 0)
		return;

	int position = hash1(keys[keyToInsert], *size);
	int key = keys[keyToInsert];
	int index = position;
	int free = 0;

	/* Check for an empty space in the HashTable */
	while(1) {
		if (pairs[index].key == key) 
			break;

		if (atomicCAS(&(pairs[index].key), free, key) == 0) {
			atomicAdd(result, 1);
			atomicAdd(numElem, 1);
			break;
		}

		index++;
		if (index == (*size))
			index = 0;
	}

	pairs[index].value = values[keyToInsert];
}

/* INSERT BATCH
 */
bool GpuHashTable::insertBatch(int *keys, int* values, int numKeys) {
	int *deviceResult, *deviceKeys, *deviceValues;
	int *hostResult;
	bool returnValue = false;

	int *hostNumElem = (int *)malloc(sizeof(int));
	hipMemcpy(hostNumElem, hashTable.numElem, sizeof(int), hipMemcpyDeviceToHost);

	int *hostSize = (int *)malloc(sizeof(int));
	hipMemcpy(hostSize, hashTable.size, sizeof(int), hipMemcpyDeviceToHost);

	/* Check if size is big enough */
	int oldSize = (*hostSize);
	while ((numKeys + (*hostNumElem)) > (*hostSize)) {
		(*hostSize) *= 2;
	}

	/* Check for hashtable density */
	if ((*hostNumElem) + numKeys > 0) { 
		if ((((float)((*hostNumElem) + numKeys) / (*hostSize)) > 0.8f) 
			&& ((*hostSize) * 2 < 0x7FFFFFFF)) {
			(*hostSize) *= 2;
		}
	}

	if (oldSize != (*hostSize))
		reshape((*hostSize));

	hostResult = (int *) malloc(sizeof(int));
	*hostResult = 0;

	hipMalloc(&deviceKeys, numKeys * sizeof(int));
	hipMalloc(&deviceValues, numKeys * sizeof(int));
	hipMalloc(&deviceResult, sizeof(int));

	hipMemcpy(deviceKeys, keys, numKeys * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(deviceValues, values, numKeys * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(deviceResult, hostResult, sizeof(int), hipMemcpyHostToDevice);

	/* Compute number of blocks */
	int blockNum = numKeys / BLOCKSIZE;
	if (blockNum * BLOCKSIZE < numKeys)
		blockNum++;

	insert<<<blockNum, BLOCKSIZE>>>(deviceKeys, deviceValues, hashTable.pairs, 
					hashTable.size, hashTable.numElem, deviceResult, numKeys);
	
	hipDeviceSynchronize();

	hipFree(deviceResult);
	hipFree(deviceKeys);
	hipFree(deviceValues);
	free(hostResult);

	return returnValue;
}

__global__ void get(int *keys, int *values, Pair *pairs, int *size, int numKeys) {
	/* Get position in HashTable for inertion */
	int keyToGet = blockIdx.x * blockDim.x + threadIdx.x;
	if (keyToGet >= numKeys)
		return;

	int position = hash1(keys[keyToGet], *size);
	int key = keys[keyToGet];
	int index = position;
	int free = 0;
	int round = 0;

	/* Check for an empty space in the HashTable */
	while ((atomicCAS(&(pairs[index].key), key, key) != key) &&
		(atomicCAS(&(pairs[index].key), free, free) != free)) {
		index++;
		if (index == (*size))
			index = 0;

		if ((index == position) && (round == 1))
			return;

		round = 1;
	}

	values[keyToGet] = pairs[index].value;
}

/* GET BATCH
 */
int* GpuHashTable::getBatch(int* keys, int numKeys) {
	int *deviceKeys, *deviceValues;
	int *hostValues;

	hostValues = (int *) calloc(numKeys, sizeof(int));

	hipMalloc(&deviceKeys, numKeys * sizeof(int));
	hipMalloc(&deviceValues, numKeys * sizeof(int));
	hipMemcpy(deviceKeys, keys, numKeys * sizeof(int), hipMemcpyHostToDevice);

	/* Compute number of blocks */
	int blockNum = numKeys / BLOCKSIZE;
	if (blockNum * BLOCKSIZE < numKeys)
		blockNum++;

	get<<<blockNum, BLOCKSIZE>>>(deviceKeys, deviceValues, hashTable.pairs, 
					hashTable.size, numKeys);

	hipDeviceSynchronize();
	hipMemcpy(hostValues, deviceValues, sizeof(int) * numKeys, hipMemcpyDeviceToHost);

	hipFree(deviceKeys);
	hipFree(deviceValues);

	return hostValues;
}

/* GET LOAD FACTOR
 * num elements / hash total slots elements
 */
float GpuHashTable::loadFactor() {
	int *numElem = (int *) malloc(sizeof(int));
	int *size = (int *) malloc(sizeof(int));

	hipMemcpy(numElem, hashTable.numElem, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(size, hashTable.size, sizeof(int), hipMemcpyDeviceToHost);

	float loadFactor = (float) *numElem / *size;

	return (float)loadFactor; // no larger than 1.0f = 100%
}

/*********************************************************/

#define HASH_INIT GpuHashTable GpuHashTable(1);
#define HASH_RESERVE(size) GpuHashTable.reshape(size);

#define HASH_BATCH_INSERT(keys, values, numKeys) GpuHashTable.insertBatch(keys, values, numKeys)
#define HASH_BATCH_GET(keys, numKeys) GpuHashTable.getBatch(keys, numKeys)

#define HASH_LOAD_FACTOR GpuHashTable.loadFactor()

#include "test_map.cpp"
